#include "hip/hip_runtime.h"
#ifndef SEARCH_RESULTS
#define SEARCH_RESULTS 4
#endif

typedef struct {
    uint32_t count;
    struct {
        // One word for gid and 4 for mix hash
        uint32_t gid;
        uint32_t mix[4];
    } result[SEARCH_RESULTS];
} search_results;

typedef struct
{
    uint32_t uint32s[32 / sizeof(uint32_t)];
} hash32_t;

// Implementation based on:
// https://github.com/mjosaarinen/tiny_sha3/blob/master/sha3.c
// converted from 64->32 bit words

__device__ __constant__ const uint32_t keccakf_rndc[24] = {
    0x00000001, 0x00008082, 0x0000808a, 0x80008000, 0x0000808b, 0x80000001,
    0x80008081, 0x00008009, 0x0000008a, 0x00000088, 0x80008009, 0x8000000a,
    0x8000808b, 0x0000008b, 0x00008089, 0x00008003, 0x00008002, 0x00000080,
    0x0000800a, 0x8000000a, 0x80008081, 0x00008080, 0x80000001, 0x80008008
};

__device__ __forceinline__ void keccak_f800_round(uint32_t st[25], const int r)
{

    const uint32_t keccakf_rotc[24] = {
        1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14,
        27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
    };
    const uint32_t keccakf_piln[24] = {
        10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4,
        15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1
    };

    uint32_t t, bc[5];
    // Theta
    for (int i = 0; i < 5; i++)
        bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

    for (int i = 0; i < 5; i++) {
        t = bc[(i + 4) % 5] ^ ROTL32(bc[(i + 1) % 5], 1);
        for (uint32_t j = 0; j < 25; j += 5)
            st[j + i] ^= t;
    }

    // Rho Pi
    t = st[1];
    for (int i = 0; i < 24; i++) {
        uint32_t j = keccakf_piln[i];
        bc[0] = st[j];
        st[j] = ROTL32(t, keccakf_rotc[i]);
        t = bc[0];
    }

    //  Chi
    for (uint32_t j = 0; j < 25; j += 5) {
        for (int i = 0; i < 5; i++)
            bc[i] = st[j + i];
        for (int i = 0; i < 5; i++)
            st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
    }

    //  Iota
    st[0] ^= keccakf_rndc[r];
}

{
    uint32_t st[25];

    for (int i = 0; i < 25; i++)
        st[i] = 0;
    for (int i = 0; i < 8; i++)
        st[i] = header.uint32s[i];
    st[8] = seed;
    st[9] = seed >> 32;
    st[10] = result.x;
    st[11] = result.y;
    st[12] = result.z;
    st[13] = result.w;

    for (int r = 0; r < 21; r++) {
        keccak_f800_round(st, r);
    }
    // last round can be simplified due to partial output
    keccak_f800_round(st, 21);

    return (uint64_t)st[1] << 32 | st[0];
}

#define fnv1a(h, d) (h = (h ^ d) * 0x1000193)

typedef struct {
    uint32_t z, w, jsr, jcong;
} kiss99_t;

// KISS99 is simple, fast, and passes the TestU01 suite
// https://en.wikipedia.org/wiki/KISS_(algorithm)
// http://www.cse.yorku.ca/~oz/marsaglia-rng.html
__device__ __forceinline__ uint32_t kiss99(kiss99_t &st)
{
    uint32_t znew = (st.z = 36969 * (st.z & 65535) + (st.z >> 16));
    uint32_t wnew = (st.w = 18000 * (st.w & 65535) + (st.w >> 16));
    uint32_t MWC = ((znew << 16) + wnew);
    uint32_t SHR3 = (st.jsr ^= (st.jsr << 17), st.jsr ^= (st.jsr >> 13), st.jsr ^= (st.jsr << 5));
    uint32_t CONG = (st.jcong = 69069 * st.jcong + 1234567);
    return ((MWC^CONG) + SHR3);
}

__device__ __forceinline__ void fill_mix(uint64_t seed, uint32_t lane_id, uint32_t mix[PROGPOW_REGS])
{
    // Use FNV to expand the per-warp seed to per-lane
    // Use KISS to expand the per-lane seed to fill mix
    uint32_t fnv_hash = 0x811c9dc5;
    kiss99_t st;
    st.z = fnv1a(fnv_hash, seed);
    st.w = fnv1a(fnv_hash, seed >> 32);
    st.jsr = fnv1a(fnv_hash, lane_id);
    st.jcong = fnv1a(fnv_hash, lane_id);
    #pragma unroll
    for (int i = 0; i < PROGPOW_REGS; i++)
        mix[i] = kiss99(st);
}

__global__ void progpow_gpu_hash(uint64_t start_nonce, const hash32_t header, const uint64_t target, const uint64_t *g_dag, volatile search_results *g_output)
{
	__shared__ uint32_t c_dag[PROGPOW_CACHE_WORDS];
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t const nonce = start_nonce + gid;

    const uint32_t lane_id = threadIdx.x & (PROGPOW_LANES - 1);

    // Load random data into the cache
    // TODO: should be a new blob of data, not existing DAG data
    for (uint32_t word = threadIdx.x*2; word < PROGPOW_CACHE_WORDS; word += blockDim.x*2)
    {
        uint64_t data = g_dag[word];
        c_dag[word + 0] = data;
        c_dag[word + 1] = data >> 32;
    }

    uint4 result;
    result.x = result.y = result.z = result.w = 0;
    // keccak(header..nonce)
    uint64_t seed = keccak_f800(header, nonce, result);

    __syncthreads();

    #pragma unroll 1
    for (uint32_t h = 0; h < PROGPOW_LANES; h++)
    {
        uint32_t mix[PROGPOW_REGS];

        // share the hash's seed across all lanes
        uint64_t hash_seed = __shfl_sync(0xFFFFFFFF, seed, h, PROGPOW_LANES);
        // initialize mix for all lanes
        fill_mix(hash_seed, lane_id, mix);

        #pragma unroll 1
        for (uint32_t l = 0; l < PROGPOW_CNT_MEM; l++)
            progPowLoop(l, mix, g_dag, c_dag);


        // Reduce mix data to a single per-lane result
        uint32_t mix_hash = 0x811c9dc5;
        #pragma unroll
        for (int i = 0; i < PROGPOW_REGS; i++)
            fnv1a(mix_hash, mix[i]);

        // Reduce all lanes to a single 128-bit result
        uint4 result_hash;
        result_hash.x = result_hash.y = result_hash.z = result_hash.w = 0x811c9dc5;
        #pragma unroll
        for (int i = 0; i < PROGPOW_LANES; i += 4)
        {
            fnv1a(result_hash.x, __shfl_sync(0xFFFFFFFF, mix_hash, i + 0, PROGPOW_LANES));
            fnv1a(result_hash.y, __shfl_sync(0xFFFFFFFF, mix_hash, i + 1, PROGPOW_LANES));
            fnv1a(result_hash.z, __shfl_sync(0xFFFFFFFF, mix_hash, i + 2, PROGPOW_LANES));
            fnv1a(result_hash.w, __shfl_sync(0xFFFFFFFF, mix_hash, i + 3, PROGPOW_LANES));
        }
        if (h == lane_id)
            result = result_hash;
    }

    // keccak(header .. keccak(header..nonce) .. result);
    if (keccak_f800(header, seed, result) > target)
        return;

    uint32_t index = atomicInc((uint32_t *)&g_output->count, 0xffffffff);
    if (index >= SEARCH_RESULTS)
        return;

    g_output->result[index].gid = gid;
    g_output->result[index].mix[0] = result.x;
    g_output->result[index].mix[1] = result.y;
    g_output->result[index].mix[2] = result.z;
    g_output->result[index].mix[3] = result.w;
}
